
#include <hip/hip_runtime.h>

#include "const.h"
#include "kernels.h"
#include "csvloader.h"

#define DEBUG

/* TODO: Error checking */
/* TODO: Better output formatting */

int main(int argc, char **argv) {

  char *fn;
  int x, y;
  FILE *csv;
  COLUMN_TYPE *arr;

  fn = argv[1];

  /* Open CSV file */
  if (fn == NULL) {
    fprintf(stderr, "Error: no file name supplied.\n\tUsage %s <input_csv_filename>\n", args[0]);
    return EXIT_FAILURE;
  }
  csv = fopen(fn, "r");
  if (csv == NULL) {
    fprintf(stderr, "Failed to open file %s\n", fn);
    return EXIT_FAILURE;
  }

  /* Read CSV file data into memory */
  x = y = 0;
  if (rowct(csv, &y) == EXIT_FAILURE || colct(csv, &x) == EXIT_FAILURE)
    return EXIT_FAILURE;
  if (readcsv(csv, x, y, &arr) == EXIT_FAILURE)
    return EXIT_FAILURE;
  fclose(csv);

  /* Set CUDA Thread / Block Limits */
  int tpb = 128;
  int bpg_multi = (y + tpb - 1) / tpb;
  int bpg_singl = ((x * y) + tpb - 1) / tpb;

  size_t ct_size = sizeof(COLUMN_TYPE);

  /* Copy the input array over to the device */
  COLUMN_TYPE *d_arr = NULL;
  if (hipMalloc((void **)&d_arr, x * y * ct_size) != hipSuccess) {
    return EXIT_FAILURE;
  }
  if (hipMemcpy(d_arr, arr, x * y * ct_size,
      hipMemcpyHostToDevice) != hipSuccess) {
    return EXIT_FAILURE;
  }

  /* Perform the signalMagnitude calculation */
  COLUMN_TYPE *d_mag = NULL, *mag;
  hipMalloc((void **)&d_mag, y * ct_size);
  signalMagnitude<<<bpg_multi, tpb>>>(d_mag, d_arr, x, y);
  mag = (COLUMN_TYPE *)calloc(y, ct_size);
  hipMemcpy(mag, d_mag, y * ct_size, hipMemcpyDeviceToHost);
  hipFree(d_mag);

  /* Perform the averageMovementIntensity calculation */
  COLUMN_TYPE *d_ami = NULL, *ami = NULL;
  hipMalloc((void **)&d_ami, y * ct_size);
  averageMovementIntensity<<<bpg_multi, tpb>>>(d_ami, d_arr, x, y);
  ami = (COLUMN_TYPE *)calloc(y, ct_size);
  hipMemcpy(ami, d_ami, y * ct_size, hipMemcpyDeviceToHost);
  hipFree(d_ami);

  /* Perform the standardDeviation / mean calculation */
  COLUMN_TYPE *d_dev = NULL, *d_avg = NULL, *dev = NULL, *avg = NULL;
  hipMalloc((void **)&d_dev, x * y * ct_size);
  hipMalloc((void **)&d_avg, x * y * ct_size);
  standardDeviation<<<bpg_singl, tpb>>>(d_dev, d_avg, d_arr, x, y, x * y);
  dev = (COLUMN_TYPE *)calloc(x * y, ct_size);
  avg = (COLUMN_TYPE *)calloc(x * y, ct_size);
  hipMemcpy(dev, d_dev, x * y * ct_size, hipMemcpyDeviceToHost);
  hipMemcpy(avg, d_avg, x * y * ct_size, hipMemcpyDeviceToHost);
  hipFree(d_dev);
  hipFree(d_avg);

  /* Complete usage of the data array */
  hipFree(d_arr);

  /* Output the results */
  fprintf(stdout, "ID");
  for (int r = 1; r <= x; r++)
    fprintf(stdout, ",INPUT_%d", r);
  fprintf(stdout, ",MAG,AMI");
  for (int r = 1; r <= x; r++)
    fprintf(stdout, ",STDEV_%d", r);
  for (int r = 1; r <= x; r++)
    fprintf(stdout, ",MEAN_%d", r);
  fprintf(stdout, "\n");
  for (int q = 0; q < y - WINDOW; q++) {
    fprintf(stdout, "%d", q);
    for (int r = 0; r < x; r++)
      fprintf(stdout, OUT_FORMAT_READING, arr[(q * x) + r]);
    fprintf(stdout, OUT_FORMAT_MAG, mag[q]);
    fprintf(stdout, OUT_FORMAT_AMI, ami[q]);
    for (int r = 0; r < x; r++)
      fprintf(stdout, OUT_FORMAT_STD, dev[(q * x) + r]);
    for (int r = 0; r < x; r++)
      fprintf(stdout, OUT_FORMAT_AVG, avg[(q * x) + r]);
    fprintf(stdout, "\n");
  }

  /* Release all memory */
  free(arr);
  free(mag);
  free(ami);
  free(dev);
  free(avg);

  return EXIT_SUCCESS;

}
