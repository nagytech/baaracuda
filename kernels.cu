#include "hip/hip_runtime.h"
/*
 * kernels.cu
 * ----------
 *
 * Author: Jonathan Nagy <jnagy@myune.edu.au>
 * Date:   18 Sep 2016
 * Description:
 *
 *    This file contains a set of CUDA kernels used for computing various
 * statistical properties.  Note that all values are calculated within a
 * sliding window of `n` to `n + (WINDOW - 1)`.
 *
 *    All kernels are capable of accepting any number of input values and will
 * collapse the output into a single value.  However, some kernels are designed
 * to compute values _across_ features, while others are designed to compute
 * _within_ features.  This is determined by the presence of a precalculated
 * size (written as xy) which depticts the complete dimensions of the array and
 * indicates that each individual feature is to be used as an input.
 *
 * ------------------------------------------------------------------------ */

#include "const.h"
#include "kernels.h"

/* Function definitions */
#define ABS_FUNC                    fabs    /* For computing absolute value */
#define SQRT_FUNC                   sqrtf   /* For computing square root */

/*
 * signalMagnitude
 * ---------------
 * Calculates the signal magnitude across all input features
 *
 * Math TeX 3 feature example:
 *
 *   SMA =
 *     \frac{1}{T}\big(
 *       \sum\limits_{i=1}^T |a_x(i)| +
 *       \sum\limits_{i=1}^T |a_y(i)| +
 *       \sum\limits_{i=1}^T |a_z(i)|
 *     \big)
 *
 * ans:   array of output values, will be of length y
 * arr:   array of input values, stored in row major order by rank (x, y)
 * x:     width of data array
 * y:     height of data array
 *
 */
__global__ void signalMagnitude(DATA_T *ans, const DATA_T *arr, int x, int y) {

  /* Derive data index */
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int t = threadIdx.x;

  /* Block shared memory */
  extern __shared__ DATA_T sh[];

  /* Filter over-allocated threads, but ignore last (window - 1) records */
  if (i + WINDOW < y) {

    int j, k;
    DATA_T sig = 0;

    /* Set local prior to calcs */
#pragma unroll
    for (k = 0; k < x; k++) {
      sh[t] = ABS_FUNC(arr[(x * i) + k]);
    }

    /* Overfill shared memory to account for overflowing window calcs */
    if (t == 0)
#pragma unroll
    for (k = 0; k < x; k++) {
#pragma unroll
      for (j = 0; j < WINDOW; j++) {
        sh[TPB + j] = ABS_FUNC(arr[(x * (TPB + j)) + k]);
      }
    }

    /* Wait for thread 0 to finish with overflow */
    __syncthreads();

    /* Iterate through columns within current window position */
#pragma unroll
    for (k = 0; k < x; k++) {
      /* Iterate through sliding window */
#pragma unroll
      for (j = 0; j < WINDOW; j++) {
        /* Accumulate the signal strength */
        sig += sh[(x * (t + j)) + k];
      }
    }

    /* Mean all values */
    ans[i] = sig / WINDOW;

  } else if (i < y) {
    /* Avoid a NULL reference */
    ans[i] = 0;
  }
}

/*
 * averageMovementIntensity
 * ------------------------
 * Calculates the average movement intensity across all input features
 *
 * Math TeX 3 feature example:
 *
 *   MI_{avg} = \frac{1}{T}\big(
 *      \sum\limits_{i=1}^T (
 *        a_x(i)^2 + a_y(i)^2)+a_z(i)^2
 *      )
 *   \big)
 *
 * ans:   array of output values, will be of length y
 * arr:   array of input values, stored in row major order by rank (x, y)
 * x:     width of data array
 * y:     height of data array
 *
 */
__global__ void averageMovementIntensity(DATA_T *ans, const DATA_T *arr,
  int x, int y) {

  /* Derive data index */
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int t = threadIdx.x;

  /* Block shared memory */
  extern __shared__ DATA_T sh[];

  /* Filter over-allocated threads, but ignore last (window - 1) records */
  if (i + WINDOW < y) {

    int j, k;

    /* Set local prior to calcs */
#pragma unroll
    for (k = 0; k < x; k++) {
      sh[t] = arr[(x * i) + k] * arr[(x * i) + k];
    }

    /* Overfill shared memory to account for overflowing window calcs */
    if (t == 0)
#pragma unroll
    for (k = 0; k < x; k++) {
#pragma unroll
      for (j = 0; j < WINDOW; j++) {
        sh[TPB + j] = arr[(x * (TPB + j)) + k] * arr[(x * (TPB + j)) + k];
      }
    }

    /* Wait for thread 0 to finish with overflow */
    __syncthreads();

    DATA_T sig = 0;

    /* Iterate through columns within current window position */
#pragma unroll
    for (k = 0; k < x; k++) {
      /* Iterate through sliding window */
#pragma unroll
      for (j = 0; j < WINDOW; j++) {
        /* Accumulate the intensity value */
        sig += sh[(x * (t + j)) + k];
      }
    }

    /* Mean all values */
    ans[i] = sig / WINDOW;

  } else if (i < y) {
    /* Avoid a null reference */
    ans[i] = 0;
  }

}

/*
 * standardDeviation
 * -----------------
 * Calculates the standard deviaion _and_ mean _for each_ input feature
 * across all input features.
 *
 * Math TeX example:
 *
 * sd_x = \sqrt{\sum\limits_{i=1}^T (a_x(i) - \bar{a_x})^2}
 * ax_{average} = \frac{1}{T}\big(\sum\limits_{i=1}^T (a_x(i) \big)
 *
 * dev:   array of output values, will be of length (y * x) and indexed by
          row major order rank (x, y) - (standard deviation)
 * avg:   array of output values, will be of length (y * x) and indexed by
          row major order rank (x, y) - (mean)
 * arr:   array of input values, stored in row major order by rank (x, y)
 * x:     width of data array
 * y:     height of data array
 *
 */
__global__ void standardDeviation(DATA_T *dev, DATA_T *avg, const DATA_T *arr,
  int x, int y, int xy) {

  /* Derive data index */
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int t = threadIdx.x;

  /* Block shared memory */
  extern __shared__ DATA_T sh[];

  /* Filter over-allocated threads, but ignore last (window - 1) records */
  if (i + (WINDOW * x) < xy) {

    int j, k;

    /* Set local prior to calcs */
#pragma unroll
    for (k = 0; k < x; k++) {
      sh[t] = arr[(x * i) + k];
    }

    /* Overfill shared memory to account for overflowing window calcs */
    if (t == 0)
#pragma unroll
    for (j = 0; j < WINDOW; j++) {
#pragma unroll
      for (k = 0; k < x; k++) {
        sh[TPB + j] = arr[(x * (TPB + j)) + k];
      }
    }

    /* Wait for thread 0 to finish with overflow */
    __syncthreads();

    DATA_T mean, sig, sum;
    sum = 0; sig = 0;

    /* Iterate through sliding window for summation */
#pragma unroll
    for (j = 0; j < WINDOW; j++)
      sum += sh[t + (j * x)];

    mean = sum / WINDOW;

    /* Iterate through sliding for standard deviation */
#pragma unroll
    for (int j = 0; j < WINDOW; j++)
      sig += sh[t + (j * x)] - mean;

    /* Calculate standard deviation */
    sig *= sig;
    sig /= WINDOW;

    avg[i] = mean;
    dev[i] = SQRT_FUNC(sig);

  } else if (i < xy) {
    /* Avoid NULL reference */
    avg[i] = 0;
    dev[i] = 0;
  }
  
}

/**
 * minmax
 * ------
 * Calculates the minimum and maximum values for each feature of the
 * input dataset.
 *
 * @param min array of output values representing minimum value of sliding
 * window (row major order)
 * @param max array of output values representign maximum value of sliding
 * window (column major order)
 * @param arr input array, row major order
 * @param x   width of data
 * @param y   length of data
 */
__global__
void minmax(DATA_T *min, DATA_T *max, const DATA_T *arr, int x, int y, int xy) {

    /* Derive data indexes */
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int t = threadIdx.x;

    /* Block shared memory */
    extern __shared__ DATA_T sh[];

    /* Filter over-allocated threads, but ignore last (window - 1) records */
    if (i + (WINDOW * x) < xy) {

      int j, k;
      j = 0;

      /* Set local prior to calcs */
#pragma unroll
      for (k = 0; k < x; k++) {
        sh[t] = arr[(x * i) + k];
      }

      /* Overfill shared memory to account for overflowing window calcs */
      if (t == 0)
#pragma unroll
      for (j = 0; j < WINDOW; j++) {
#pragma unroll
        for (k = 0; k < x; k++) {
          sh[TPB + j] = arr[(x * (TPB + j)) + k];
        }
      }

      /* Wait for thread 0 to finish overloading */
      __syncthreads();

      DATA_T val, lmin, lmax;
      val = lmin = lmax = sh[t + (j * x)];

#pragma unroll
      for (j = 1; j <= WINDOW; j++) {

        lmin = fminf(lmin, val);
        lmax = fmaxf(lmax, val);

        val = sh[t + (j * x)];

      }

      min[i] = lmin;
      max[i] = lmax;

    }

  }
